#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define D 2         // nb of coords
#define K 10        // Number of clusters
#define TPB 32      // Number of threads per block

// Euclidean distance of two 2D points
__device__ float distance(float x1, float y1, float x2, float y2) {
    return sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1));
}

__global__ void kMeansClusterAssignment(float* d_datapoints, int* d_clust_assn, float* d_centroids, int N) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    float min_dist = FLT_MAX;
    int closest_centroid = -1;

    for (int c = 0; c < K; ++c) {
        float dist = distance(d_datapoints[2 * idx], d_datapoints[2 * idx + 1], d_centroids[2 * c], d_centroids[2 * c + 1]);
        if (dist < min_dist)
        {
            min_dist = dist;
            closest_centroid = c;
        }
    }
    d_clust_assn[idx] = closest_centroid;
}

__global__ void kMeansCentroidUpdate(float* h_datapoints, int* h_clust_assn, float* h_centroids, int* h_clust_sizes, int N, int k) {

    float clust_datapoint_sums[2 * K] = { 0 };

    for (int j = 0; j < N; ++j) {
        int clust_id = h_clust_assn[j];
        clust_datapoint_sums[2 * clust_id] += h_datapoints[2 * j];
        clust_datapoint_sums[2 * clust_id + 1] += h_datapoints[2 * j + 1];
        h_clust_sizes[clust_id] += 1;
    }
    for (int idx = 0; idx < K; idx++) {
        if (h_clust_sizes[idx])
        {
            h_centroids[2 * idx] = clust_datapoint_sums[2 * idx] / h_clust_sizes[idx];
            h_centroids[2 * idx + 1] = clust_datapoint_sums[2 * idx + 1] / h_clust_sizes[idx];
        }
    }

}

void read_points_from_file(const char *filename, float *h_datapoints, int num_points) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Unable to open file %s.\n", filename);
        exit(1);
    }
    for (int i = 0; i < num_points; i++) {
        float x, y;
        if (fscanf(file, "%f %f", &x, &y) != 2) {
            fprintf(stderr, "Error reading from file.\n");
            exit(1);
        }
        h_datapoints[2 * i] = x;
        h_datapoints[2 * i + 1] = y;
    }
    fclose(file);
}

void centroid_init(float* h_datapoints, float* h_centroids, int N) {
	for (int i = 0; i < K; i++) {
		int temp = (N / K);
		int idx_r = rand() % temp;
		h_centroids[2 * i] = h_datapoints[(i * temp + idx_r)];
		h_centroids[2 * i + 1] = h_datapoints[(i * temp + idx_r) + 1];
	}
}

int main() {
    const char* filename;
    int N=400000, MAX_ITER;
    filename = "points_250_000.txt";
    MAX_ITER = 4000;
    float* d_datapoints = 0;
    int* d_clust_assn = 0;
    float* d_centroids = 0;
    int* d_clust_sizes = 0;

    hipMalloc(&d_datapoints, D * N * sizeof(float));
    hipMalloc(&d_clust_assn, N * sizeof(int));
    hipMalloc(&d_centroids, D * K * sizeof(float));
    hipMalloc(&d_clust_sizes, K * sizeof(float));

    float* h_centroids = (float*)malloc(D * K * sizeof(float));
    float* h_datapoints = (float*)malloc(D * N * sizeof(float));
    int* h_clust_sizes = (int*)malloc(K * sizeof(int));
    int* h_clust_assn = (int*)malloc(N * sizeof(int));

    read_points_from_file(filename, h_datapoints, N);

    centroid_init(h_datapoints, h_centroids, N);

    for (int c = 0; c < K; ++c) {
        h_clust_sizes[c] = 0;
    }

    clock_t start_time = clock();

    for (int cur_iter = 0; cur_iter < MAX_ITER; cur_iter++) {
        kMeansClusterAssignment<<<(N + TPB - 1) / TPB, TPB>>>(d_datapoints, d_clust_assn, d_centroids, N);

        kMeansCentroidUpdate<<<(K + TPB - 1) / TPB, TPB>>>(d_datapoints, d_clust_assn, d_centroids, d_clust_sizes, N, K);

        if (cur_iter == MAX_ITER - 1) {
            hipMemcpy(h_centroids, d_centroids, D * K * sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    clock_t end_time = clock();
    double execution_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    printf("Final Centroids:\n");
    for (int l = 1; l <= K; l++) {
        printf("Centroid %d: %f, %f\n", l, h_centroids[2 * l], h_centroids[2 * l + 1]);
    }

    printf("Execution Time: %f seconds\n", execution_time);

    free(h_centroids);
    free(h_datapoints);
    free(h_clust_sizes);
    free(h_clust_assn);
    hipFree(d_datapoints);
    hipFree(d_clust_assn);
    hipFree(d_centroids);

    return 0;
}
